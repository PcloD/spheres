#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"

#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable_list **world, rand_state *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if (hit_hitable_list(*world, cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (scatter(*rec.mat_ptr, cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t)*vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(rand_state *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, rand_state *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable_list **world, rand_state *rnd_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    rand_state local_rand_state = rnd_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rnd_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(sphere **d_spheres, material **d_materials, hitable_list **d_world, camera **d_camera, int nx, int ny, rand_state *rnd_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        rand_state local_rand_state = *rnd_state;
        d_materials[0] = new material(material::Lambertian, vec3(0.5, 0.5, 0.5), 0, 0);
        d_spheres[0] = new sphere(vec3(0, -1000.0, -1), 1000);
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_materials[i] = new material(material::Lambertian, vec3(RND*RND, RND*RND, RND*RND), 0, 0);
                    d_spheres[i++] = new sphere(center, 0.2);
                }
                else if (choose_mat < 0.95f) {
                    d_materials[i] = new material(material::Metal, vec3(0.5f*(1.0f + RND), 0.5f*(1.0f + RND), 0.5f*(1.0f + RND)), 0.5f*RND, 0);
                    d_spheres[i++] = new sphere(center, 0.2);
                }
                else {
                    d_materials[i] = new material(material::Dielectric, vec3(), 0, 1.5);
                    d_spheres[i++] = new sphere(center, 0.2);
                }
            }
        }
        d_materials[i] = new material(material::Dielectric, vec3(), 0, 1.5);
        d_spheres[i++] = new sphere(vec3(0, 1, 0), 1.0);
        d_materials[i] = new material(material::Lambertian, vec3(0.4, 0.2, 0.1), 0, 0);
        d_spheres[i++] = new sphere(vec3(-4, 1, 0), 1.0);
        d_materials[i] = new material(material::Metal, vec3(0.7, 0.6, 0.5), 0, 0);
        d_spheres[i++] = new sphere(vec3(4, 1, 0), 1.0);
        *rnd_state = local_rand_state;
        *d_world = new hitable_list(d_spheres, d_materials, 22 * 22 + 1 + 3);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0; (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(lookfrom,
            lookat,
            vec3(0, 1, 0),
            30.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus);
    }
}

__global__ void free_world(sphere **d_spheres, material **d_materials, hitable_list **d_world, camera **d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete d_materials[i];
        delete d_spheres[i];
    }
    delete *d_world;
    delete *d_camera;
}

void write_image(const char* output_file, const vec3 *fb, const int nx, const int ny) {
    char *data = new char[nx * ny * 3];
    int idx = 0;
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            data[idx++] = int(255.99*fb[pixel_index].r());
            data[idx++] = int(255.99*fb[pixel_index].g());
            data[idx++] = int(255.99*fb[pixel_index].b());
        }
    }
    stbi_write_png(output_file, nx, ny, 3, (void*)data, nx * 3);
    delete[] data;
}

int main() {
    int nx = 1200;
    int ny = 800;
    int ns = 10;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *d_fb;
    checkCudaErrors(hipMalloc((void **)&d_fb, fb_size));

    // allocate random state
    rand_state *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(rand_state)));
    rand_state *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1 * sizeof(rand_state)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> >(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    sphere **d_spheres;
    material **d_materials;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void **)&d_spheres, num_hitables * sizeof(sphere *)));
    checkCudaErrors(hipMalloc((void **)&d_materials, num_hitables * sizeof(material *)));
    hitable_list **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable_list *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world <<<1, 1 >>>(d_spheres, d_materials, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> >(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> >(d_fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    vec3* h_fb = new vec3[fb_size];
    checkCudaErrors(hipMemcpy(h_fb, d_fb, fb_size, hipMemcpyDeviceToHost));
    write_image("output.png", h_fb, nx, ny);
    delete[] h_fb;
    h_fb = NULL;

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world <<<1, 1 >>>(d_spheres, d_materials, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_spheres));
    checkCudaErrors(hipFree(d_materials));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_fb));

    hipDeviceReset();
}